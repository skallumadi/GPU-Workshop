#include "hip/hip_runtime.h"
/*
 *  Copyright 2014 NVIDIA Corporation
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include <stdio.h>

#ifdef DEBUG
#define CUDA_CALL(F)  if( (F) != hipSuccess ) \
  {printf("Error %s at %s:%d\n", hipGetErrorString(hipGetLastError()), \
   __FILE__,__LINE__); exit(-1);} 
#define CUDA_CHECK()  if( (hipPeekAtLastError()) != hipSuccess ) \
  {printf("Error %s at %s:%d\n", hipGetErrorString(hipGetLastError()), \
   __FILE__,__LINE__-1); exit(-1);} 
#else
#define CUDA_CALL(F) (F)
#define CUDA_CHECK() 
#endif

__global__ void add(int *a, int *b, int *c)
{
  *c = *a + *b;
}

int main()
{
  int a, b, c;
  int *d_a, *d_b, *d_c;
  int size = sizeof( int );

/* allocate space for device copies of a, b, c */

  CUDA_CALL( hipMalloc( (void **) &d_a, size ) );
/* enter code here to malloc d_b and d_c */
  FIXME

/* setup initial values */

  a = 2;
  b = 7;
  c = -99;

/* copy inputs to device */

  CUDA_CALL( hipMemcpy( d_a, &a, size, hipMemcpyHostToDevice ) );
/* enter code here to copy d_b to device */
  FIXME

/* enter code here to launch the kernel on the GPU */
  FIXME

  CUDA_CHECK()
  CUDA_CALL( hipDeviceSynchronize() );

/* copy result back to host */

  CUDA_CALL( hipMemcpy( &c, d_c, size, hipMemcpyDeviceToHost ) );

  printf("value of c after kernel is %d\n",c);
  if( c == ( a + b ) ) printf("PASS\n");
  else printf("FAIL\n");

/* clean up */

  CUDA_CALL( hipFree( d_a ) );
  FIXME
/* enter code here to hipFree the d_b and d_c pointers */

/* calling reset to check errors */
  CUDA_CALL( hipDeviceReset() );
	
  return 0;
} /* end main */
