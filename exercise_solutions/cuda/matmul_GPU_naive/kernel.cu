/*
 *  Copyright 2014 NVIDIA Corporation
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include "hipblas.h"

#ifdef DEBUG
#define CUDA_CALL(F)  if( (F) != cudaSuccess ) \
  {printf("Error %s at %s:%d\n", cudaGetErrorString(cudaGetLastError()), \
   __FILE__,__LINE__); exit(-1);} 
#define CUDA_CHECK()  if( (cudaPeekAtLastError()) != cudaSuccess ) \
  {printf("Error %s at %s:%d\n", cudaGetErrorString(cudaGetLastError()), \
   __FILE__,__LINE__-1); exit(-1);} 
#else
#define CUDA_CALL(F) (F)
#define CUDA_CHECK() 
#endif

/* macro for index calculations */

#define INDX( row, col, ld ) ( ( (col) * (ld) ) + (row) )

/* matrix size and thread dimensions */

#define SIZE 1024
#define THREADS_PER_BLOCK_X 16
#define THREADS_PER_BLOCK_Y 16

/* naive GPU kernel where each element of C is computed by a single thread */

__global__ void GPU_naive( const int m, double const * const a, double const * const b, double * const c )
{

/* determine my threads's row and col indices in the global C matrix */

  const int myrow = blockDim.x * blockIdx.x + threadIdx.x;
  const int mycol = blockDim.y * blockIdx.y + threadIdx.y;

/* if my row and col are in the C matrix, then calculate that value of C */

  if( myrow < m && mycol < m )
  {
    register double temp = 0.0;

    for( int k = 0; k < m; k++ ) 
      temp += a[INDX( myrow, k, m )] * b[INDX( k, mycol, m )];

    c[INDX( myrow, mycol, m )] = temp;
  } /* end if */

	return;
} /* end GPU_naive */

int main( int argc, char *argv[] )
{

  const int size = SIZE;

  fprintf(stdout, "Matrix size is %d\n",size);

  double *h_a, *h_b, *h_c, *h_c1;
  double *d_a, *d_b, *d_c;
 
  size_t numbytes = (size_t ) size * (size_t ) size * sizeof( double );

  h_a = (double *) malloc( numbytes );
  if( h_a == NULL )
  {
    fprintf(stderr,"Error in host malloc\n");
    return 911;
  }

  h_b = (double *) malloc( numbytes );
  if( h_b == NULL )
  {
    fprintf(stderr,"Error in host malloc\n");
    return 911;
  }

  h_c = (double *) malloc( numbytes );
  if( h_c == NULL )
  {
    fprintf(stderr,"Error in host malloc\n");
    return 911;
  }

  h_c1 = (double *) malloc( numbytes );
  if( h_c1 == NULL )
  {
    fprintf(stderr,"Error in host malloc\n");
    return 911;
  }

/* zero out the host memory for C matrices */

  memset( h_c, 0, numbytes );
  memset( h_c1, 0, numbytes );

  fprintf( stdout, "Total memory required is %lf MB\n", 
     3.0 * (double) numbytes / 1000000.0 );

/* initialize the A and B matrices */

  for( int i = 0; i < size * size; i++ )
  {
    h_a[i] = double( rand() ) / ( double(RAND_MAX) + 1.0 );
    h_b[i] = double( rand() ) / ( double(RAND_MAX) + 1.0 );
  }

/* allocate a, b, c in gpu memory */

  CUDA_CALL( hipMalloc( (void **)&d_a, numbytes ) );
  CUDA_CALL( hipMalloc( (void **)&d_b, numbytes ) );
  CUDA_CALL( hipMalloc( (void **)&d_c, numbytes ) );
	
/* copy a and b to device */

  CUDA_CALL( hipMemcpy( d_a, h_a, numbytes, hipMemcpyHostToDevice ) );
  CUDA_CALL( hipMemcpy( d_b, h_b, numbytes, hipMemcpyHostToDevice ) );

  hipblasHandle_t handle;
  hipblasCreate( &handle );

  double alpha = 1.0;
  double beta  = 0.0;

/* start timers */

  hipEvent_t start, stop;
  CUDA_CALL( hipEventCreate( &start ) );
  CUDA_CALL( hipEventCreate( &stop ) );
  CUDA_CALL( hipEventRecord( start, 0 ) );

/* call CUBLAS dgemm */

  hipblasDgemm( handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
               size, size, size,
               &alpha, 
               d_a, size,
               d_b, size,
               &beta,
               d_c, size );

/* stop timers */

  CUDA_CALL( hipEventRecord( stop, 0 ) );
  CUDA_CALL( hipEventSynchronize( stop ) );
  float elapsedTime;
  CUDA_CALL( hipEventElapsedTime( &elapsedTime, start, stop ) );

/* print GPU CUBLAS timing information */

  fprintf(stdout, "Total time GPU CUBLAS is %f sec\n", elapsedTime / 1000.0f );
  fprintf(stdout, "Performance is %f GFlop/s\n", 
    2.0 * (double) size * (double) size * (double) size / 
    ( (double) elapsedTime / 1000.0 ) * 1.e-9 );
    
/* copy C from device to host for error checking */

  CUDA_CALL( hipMemcpy( h_c, d_c, numbytes, hipMemcpyDeviceToHost ) );

/* reset C on device to zero */

  CUDA_CALL( hipMemset( d_c, 0, numbytes ) );

/* setup grid and block sizes */

  dim3 threads( THREADS_PER_BLOCK_X, THREADS_PER_BLOCK_Y, 1 );
  dim3 blocks( size / THREADS_PER_BLOCK_X + 1, 
               size / THREADS_PER_BLOCK_Y + 1, 1 );

/* start timers */

  CUDA_CALL( hipEventRecord( start, 0 ) );

/* call GPU_naive */

  GPU_naive<<< blocks, threads >>> ( size, d_a, d_b, d_c );
  CUDA_CHECK()
  CUDA_CALL( hipDeviceSynchronize() );

/* stop timers */

  CUDA_CALL( hipEventRecord( stop, 0 ) );
  CUDA_CALL( hipEventSynchronize( stop ) );
  CUDA_CALL( hipEventElapsedTime( &elapsedTime, start, stop ) );

/* print data for GPU naive */

  fprintf(stdout, "Total time GPU NAIVE is %f sec\n", elapsedTime / 1000.0f );
  fprintf(stdout, "Performance is %f GFlop/s\n", 
    2.0 * (double) size * (double) size * (double) size / 
    ( (double) elapsedTime / 1000.0 ) * 1.e-9 );
                  
/* copy C back to host */
	
  CUDA_CALL( hipMemcpy( h_c1, d_c, numbytes, hipMemcpyDeviceToHost ) );

  hipblasDestroy( handle );
  CUDA_CALL( hipEventDestroy( start ) );
  CUDA_CALL( hipEventDestroy( stop ) );

/* check CUBLAS versus GPU NAIVE numerical results */

  double temp = 0.0;

  for( int i = 0; i < size * size; i++ )
  {
    temp += ( h_c[i] - h_c1[i] ) * ( h_c[i] - h_c1[i] );
  } /* end for */

  printf("error is %f\n",temp);
  if( temp > 10 ) printf("FAIL\n");
  else printf("PASS\n");

/* cleanup */

  CUDA_CALL( hipFree( d_a ) );
  CUDA_CALL( hipFree( d_b ) );
  CUDA_CALL( hipFree( d_c ) );

  free( h_a );
  free( h_b );
  free( h_c );
  free( h_c1 );

  CUDA_CALL( hipDeviceReset() );

  return 0;
}
