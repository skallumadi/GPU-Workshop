/*
 *  Copyright 2014 NVIDIA Corporation
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */


#include <hip/hip_runtime.h>
#include <stdio.h>

#ifdef DEBUG
#define CUDA_CALL(F)  if( (F) != cudaSuccess ) \
  {printf("Error %s at %s:%d\n", cudaGetErrorString(cudaGetLastError()), \
   __FILE__,__LINE__); exit(-1);} 
#define CUDA_CHECK()  if( (cudaPeekAtLastError()) != cudaSuccess ) \
  {printf("Error %s at %s:%d\n", cudaGetErrorString(cudaGetLastError()), \
   __FILE__,__LINE__-1); exit(-1);} 
#else
#define CUDA_CALL(F) (F)
#define CUDA_CHECK() 
#endif

__global__ void add(int *a, int *b, int *c)
{
    c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

#define N 32

int main()
{
  int *a, *b, *c;
  int *d_a, *d_b, *d_c;
  int size = N * sizeof( int );

/* allocate space for device copies of a, b, c */

  CUDA_CALL( hipMalloc( (void **) &d_a, size ) );
  CUDA_CALL( hipMalloc( (void **) &d_b, size ) );
  CUDA_CALL( hipMalloc( (void **) &d_c, size ) );

/* allocate space for host copies of a, b, c and setup input values */

  a = (int *)malloc( size );
  b = (int *)malloc( size );
  c = (int *)malloc( size );

  for( int i = 0; i < N; i++ )
  {
    a[i] = b[i] = i;
    c[i] = 0;
  } /* end for */

/* copy inputs to device */

  CUDA_CALL( hipMemcpy( d_a, a, size, hipMemcpyHostToDevice ) );
  CUDA_CALL( hipMemcpy( d_b, b, size, hipMemcpyHostToDevice ) );

/* launch the kernel on the GPU */

  add<<< N, 1 >>>( d_a, d_b, d_c );
  CUDA_CHECK()
  CUDA_CALL( hipDeviceSynchronize() );

/* copy result back to host */

  CUDA_CALL( hipMemcpy( c, d_c, size, hipMemcpyDeviceToHost ) );

  for( int i = 0; i < N; i++ )
  {
    printf("c[%d] = %d\n",i,c[i]);
    if( c[i] != a[i] + b[i] )
    {
      printf("FAIL\n");
      goto end;
    } /* end if */
  } /* end for */

  printf("PASS\n");
  end:

/* clean up */

  free(a);
  free(b);
  free(c);
  CUDA_CALL( hipFree( d_a ) );
  CUDA_CALL( hipFree( d_b ) );
  CUDA_CALL( hipFree( d_c ) );
	
  return 0;
} /* end main */
