/*
 *  Copyright 2014 NVIDIA Corporation
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */


#include <hip/hip_runtime.h>
#include <stdio.h>

#ifdef DEBUG
#define CUDA_CALL(F)  if( (F) != cudaSuccess ) \
  {printf("Error %s at %s:%d\n", cudaGetErrorString(cudaGetLastError()), \
   __FILE__,__LINE__); exit(-1);} 
#define CUDA_CHECK()  if( (cudaPeekAtLastError()) != cudaSuccess ) \
  {printf("Error %s at %s:%d\n", cudaGetErrorString(cudaGetLastError()), \
   __FILE__,__LINE__-1); exit(-1);} 
#else
#define CUDA_CALL(F) (F)
#define CUDA_CHECK() 
#endif

#define N ( 1024 * 1024 )
#define RADIUS 5
#define THREADS_PER_BLOCK 64

/* stencil kernel */

__global__ void stencil_1d(int n, double *in, double *out)
{
/* allocate shared memory */
  __shared__ double temp[THREADS_PER_BLOCK + 2*(RADIUS)];

/* calculate global index in the array */
  int globalIndex = blockIdx.x * blockDim.x + threadIdx.x;
  int localIndex = threadIdx.x + RADIUS;

/* return if my global index is larger than the array size */
  if( globalIndex >= n ) return;

/* read input elements into shared memory */
  temp[localIndex] = in[globalIndex];

/* code to handle the halos.  need to make sure we don't walk off the end
   of the array */	
  if( threadIdx.x < RADIUS && globalIndex >= RADIUS )
  {
    temp[localIndex - RADIUS] = in[globalIndex - RADIUS];
  } /* end if */

  if( threadIdx.x < RADIUS && globalIndex < (n - RADIUS) )
  {
    temp[localIndex + THREADS_PER_BLOCK] = in[globalIndex + THREADS_PER_BLOCK];
  } /* end if */

  __syncthreads();
	
/* code to handle the boundary conditions */
  if( globalIndex < RADIUS || globalIndex >= (n - RADIUS) ) 
  {
    out[globalIndex] = (double) globalIndex * ( (double)RADIUS*2 + 1) ;
    return;
  } /* end if */

  double result = 0.0;

  for( int i = -(RADIUS); i <= (RADIUS); i++ ) 
  {
    result += temp[localIndex + i];
  } /* end for */

  out[globalIndex] = result;
  return;

}

int main()
{
  double *in, *out;
  double *d_in, *d_out;
  int size = N * sizeof( double );

/* allocate space for device copies of in, out */

  CUDA_CALL( hipMalloc( (void **) &d_in, size ) );
  CUDA_CALL( hipMalloc( (void **) &d_out, size ) );

/* allocate space for host copies of in, out and setup input values */

  in = (double *)malloc( size );
  out = (double *)malloc( size );

  for( int i = 0; i < N; i++ )
  {
    in[i] = (double) i;
    out[i] = -99.0;
  }

/* copy inputs to device */

  CUDA_CALL( hipMemcpy( d_in, in, size, hipMemcpyHostToDevice ) );
  CUDA_CALL( hipMemset( d_out, 0, size ) );

/* calculate block and grid sizes */

  dim3 threads( THREADS_PER_BLOCK, 1, 1);
  dim3 blocks( (N / threads.x) + 1, 1, 1);

/* start the timers */

  hipEvent_t start, stop;
  CUDA_CALL( hipEventCreate( &start ) );
  CUDA_CALL( hipEventCreate( &stop ) );
  CUDA_CALL( hipEventRecord( start, 0 ) );

/* launch the kernel on the GPU */

  stencil_1d<<< blocks, threads >>>( N, d_in, d_out );
  CUDA_CHECK();
  CUDA_CALL( hipDeviceSynchronize() );

/* stop the timers */

  CUDA_CALL( hipEventRecord( stop, 0 ) );
  CUDA_CALL( hipEventSynchronize( stop ) );
  float elapsedTime;
  CUDA_CALL( hipEventElapsedTime( &elapsedTime, start, stop ) );

  printf("Total time for %d elements was %f ms\n", N, elapsedTime );

/* copy result back to host */

  CUDA_CALL( hipMemcpy( out, d_out, size, hipMemcpyDeviceToHost ) );

  for( int i = 0; i < N; i++ )
  {
    if( in[i]*( (double)RADIUS*2+1 ) != out[i] ) 
    {
      printf("error in element %d in = %f out %f\n",i,in[i],out[i] );
      printf("FAIL\n");
      goto end;
    } /* end if */
  } /* end for */

  printf("PASS\n");
  end:

/* clean up */

  free(in);
  free(out);
  CUDA_CALL( hipFree( d_in ) );
  CUDA_CALL( hipFree( d_out ) );

  CUDA_CALL( hipDeviceSynchronize() );
	
  return 0;
} /* end main */
