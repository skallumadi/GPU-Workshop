/*
 *  Copyright 2014 NVIDIA Corporation
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */


#include <hip/hip_runtime.h>
#include <stdio.h>

#ifdef DEBUG
#define CUDA_CALL(F)  if( (F) != cudaSuccess ) \
  {printf("Error %s at %s:%d\n", cudaGetErrorString(cudaGetLastError()), \
   __FILE__,__LINE__); exit(-1);} 
#define CUDA_CHECK()  if( (cudaPeekAtLastError()) != cudaSuccess ) \
  {printf("Error %s at %s:%d\n", cudaGetErrorString(cudaGetLastError()), \
   __FILE__,__LINE__-1); exit(-1);} 
#else
#define CUDA_CALL(F) (F)
#define CUDA_CHECK() 
#endif

__global__ void add(int *a, int *b, int *c)
{
  *c = *a + *b;
}

int main()
{
  int a, b, c;
  int *d_a, *d_b, *d_c;
  int size = sizeof( int );

/* allocate space for device copies of a, b, c */

  CUDA_CALL( hipMalloc( (void **) &d_a, size ) );
  CUDA_CALL( hipMalloc( (void **) &d_b, size ) );
  CUDA_CALL( hipMalloc( (void **) &d_c, size ) );

/* setup initial values */

  a = 2;
  b = 7;
  c = -99;

/* copy inputs to device */

  CUDA_CALL( hipMemcpy( d_a, &a, size, hipMemcpyHostToDevice ) );
  CUDA_CALL( hipMemcpy( d_b, &b, size, hipMemcpyHostToDevice ) );

/* launch the kernel on the GPU */

  add<<< 1, 1 >>>( d_a, d_b, d_c );
  CUDA_CHECK()
  CUDA_CALL( hipDeviceSynchronize() );

/* copy result back to host */

  CUDA_CALL( hipMemcpy( &c, d_c, size, hipMemcpyDeviceToHost ) );

  printf("value of c after kernel is %d\n",c);
  if( c == ( a + b ) ) printf("PASS\n");
  else printf("FAIL\n");

/* clean up */

  CUDA_CALL( hipFree( d_a ) );
  CUDA_CALL( hipFree( d_b ) );
  CUDA_CALL( hipFree( d_c ) );

  CUDA_CALL( hipDeviceReset() );
	
  return 0;
} /* end main */
